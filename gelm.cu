
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <iomanip>

using namespace std;

void rowOperation(float** __restrict__ coefMatrix, float* __restrict__ constMatrix, int row1, int row2, int size)
{
    float scalar = coefMatrix[row1][row2] / coefMatrix[row2][row2];
    for(int i = 0; i < size; i++)
    {
        coefMatrix[row1][i] -= coefMatrix[row2][i] * scalar;
    }
    constMatrix[row1] -= constMatrix[row2] * scalar;
}

bool gaussElimination(float** __restrict__ coefMatrix, float* __restrict__ constMatrix, int size)
{
    for(int row = 0; row < size; row++)
    {
        for(int rowAfterLeadingOne = row + 1; rowAfterLeadingOne < size; rowAfterLeadingOne++)
        {
            rowOperation(coefMatrix, constMatrix, rowAfterLeadingOne, row, size);
        }
    }
    return true;
}



void PrintResult(float** __restrict__ coefMatrix, float* __restrict__ constMatrix, int size)
{
    float result[size];

    result[size - 1] = constMatrix[size - 1] / coefMatrix[size - 1][size - 1];

    for(int i = size - 2; i >= 0; i--)
    {
        result[i] = constMatrix[i];
        for (int j = i + 1; j < size; j++)
            result[i] -= coefMatrix[i][j] * result[j];
        result[i] /= coefMatrix[i][i];
    }

    cout << "Result : (";

    for (int i = 0; i < size; i++)
    {
        cout << result[i];

        if(i < size - 1)
            cout << ", ";
    }

    cout << ")" << endl;

}


void CopyMatrixFromFile(float* matrix, ifstream& inFile, int size)
{
    for(int i = 0; i < size; i++)
        inFile >> matrix[i];
}

void PrintMatrix(float* matrix, int sizeOfMatrix, int size)
{
    cout << endl;
    int modValue = sizeOfMatrix;

    for(int i = 0; i < size; i++)
    {
        cout << setprecision(3) << matrix[i] << " \t";
        if(i % modValue == 0 && i != 0)
        {
        	modValue += (sizeOfMatrix + 1);
			cout << endl << endl << endl;
        }
    }
}

__global__ void gelm(float* __restrict__ d_matrix, float* __restrict__ d_result, const int sizeOfMatrix, const int size)
{
    const int index = threadIdx.x + blockIdx.x * blockDim.x;

    if(index < size)
    {
        const int sizeOfMatrixPlus1 = sizeOfMatrix + 1;
        const int sizeOfMatrixPlus2 = sizeOfMatrixPlus1 + 1;
        const int sizeOfMatrixMinus1 = sizeOfMatrix - 1;
        const int sizeOfMatrixPlus2IntoIndex = sizeOfMatrixPlus2 * index;
        int rowIntoSizeOfMatrixPlus1PlusPointer;
        int lead;
        int i;
        for(i = 0; i < size; i += sizeOfMatrixPlus2)
        {
            if(index > (i + sizeOfMatrix))
            {
                rowIntoSizeOfMatrixPlus1PlusPointer = ((index / sizeOfMatrixPlus1) * sizeOfMatrixPlus1) + (i / sizeOfMatrixPlus1);
                d_matrix[index] -= d_matrix[index + i - rowIntoSizeOfMatrixPlus1PlusPointer] * (d_matrix[rowIntoSizeOfMatrixPlus1PlusPointer] / d_matrix[i]);
            }
        }
        __syncthreads();
        // INITIALIZATION
        if(index < sizeOfMatrix)
        {
            lead = (size - 1) - (sizeOfMatrixPlus1 * (sizeOfMatrixMinus1 - index));
            d_result[index] = d_matrix[lead] / d_matrix[lead - sizeOfMatrix + index];
			__syncthreads();
			// BACK SUBSTITUTION PROCESS
			for(i = sizeOfMatrixMinus1; i > 0; i--)
			{
				if(index < i)
					d_result[index] -= ((d_matrix[sizeOfMatrixPlus2IntoIndex + (i - index)] / d_matrix[sizeOfMatrixPlus2IntoIndex]) * d_result[i]);
			}
			__syncthreads();
        }
    }
}

int main()
{
    float* matrix;
    float* d_matrix;
    float* result;
    float* d_result;

    int sizeOfMatrix;
    ifstream inFile("inputMatrix.txt");
    inFile >> sizeOfMatrix;

    int size = sizeOfMatrix * (sizeOfMatrix + 1);

    float sizeInBytes = size * sizeof(float);

    matrix = new float [size];
    result = new float [sizeOfMatrix];

    CopyMatrixFromFile(matrix, inFile, size);

    float** coefMatrix;
    float* constMatrix;
    coefMatrix = new float* [sizeOfMatrix];
    constMatrix = new float [sizeOfMatrix];

    for(int i = 0; i < sizeOfMatrix; i++)
        coefMatrix[i] = new float [sizeOfMatrix];

    for(int row = 0; row < sizeOfMatrix; row++)
    {
        for(int column = 0; column < sizeOfMatrix; column++)
        {
            coefMatrix[row][column] = matrix[column + row * (sizeOfMatrix + 1)];

            if(column == sizeOfMatrix - 1)
                constMatrix[row] = matrix[(column + 1) + row * (sizeOfMatrix + 1)];
        }
    }

    float start_s = clock();

    bool uniqueSoln = gaussElimination(coefMatrix, constMatrix, sizeOfMatrix);
    if(uniqueSoln)
    	PrintResult(coefMatrix, constMatrix, sizeOfMatrix);

    float stop_s = clock();

    float cpu = (stop_s-start_s)/float(CLOCKS_PER_SEC)*1000;
    cout << "time CPU: " << cpu << endl;

    hipMalloc(&d_matrix, sizeInBytes);
    hipMalloc(&d_result, (sizeOfMatrix * sizeof(float)));

    start_s = clock();

    hipMemcpy(d_matrix, matrix, sizeInBytes, hipMemcpyHostToDevice);

    gelm<<<size / 1024 + 1, 1024/*, sizeOfMatrix + 1*/>>>(d_matrix, d_result, sizeOfMatrix, size);

    hipMemcpy(result, d_result, (sizeOfMatrix * sizeof(float)), hipMemcpyDeviceToHost);

    cout << "\nResult from GPU : (";

    for (int i = 0; i < sizeOfMatrix; i++)
    {
        cout << result[i];

        if(i < sizeOfMatrix - 1)
            cout << ", ";
    }

    cout << ")" << endl;

    stop_s = clock();

    float gpu = (stop_s-start_s)/float(CLOCKS_PER_SEC)*1000;
    cout << "time GPU: " << gpu << endl;

    cout << "Improvement: " << setprecision(4) << (cpu - gpu) / cpu << endl;
    cout << "Improvement: " << setprecision(4) << cpu / gpu << endl;

        // ****************** //

    cout << "\n";

    for(int i = 0; i < sizeOfMatrix; i++)
        delete[] coefMatrix[i];

    delete[] coefMatrix;

    delete[] constMatrix;

        // *********** //

    hipFree(d_matrix);
    hipFree(d_result);
    delete[] matrix;
    delete[] result;
    return 0;
}

